#include <stdlib.h>
#include <stdio.h>
#include <stdexcept>
#include <cstdlib>
#include <hip/hip_runtime.h>
#include <iostream>
#include <math.h>
#include <sys/time.h>
#include "hipblas.h"

#ifndef N
#define N 32
#endif

using namespace std;

void lib_mmul(float* A, float* B, float* C) {
	
	const float alf = 1;
	const float bet = 0;
	const float *alpha = &alf;
	const float *beta = &bet;

	hipblasHandle_t handle;
	hipblasCreate(&handle);

	//switch A and B because this function uses column major order
  hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, N, N, N, alpha, B, N, A, N, beta, C, N);

	hipblasDestroy(handle);

}

int main(){

	size_t size = N * N * sizeof(float);
	struct timeval start_c, end_c, start_t, end_t;
	double time_total, time_calc;

	//host declaration and memory reservation
	float* h_A = (float*)malloc(size);
	float* h_B = (float*)malloc(size);
	float* h_C = (float*)malloc(size);

	//initialize arrays
	int i,j;
  for(i = 0; i < N; i++){
    for(j = 0; j < N; j++){
      h_A[N*i+j] = (float)(i);
      h_B[N*i+j] = (float)(i+j);
    }
  }

  //device initialization and memory allocation
	float *d_A, *d_B, *d_C;

	gettimeofday(&start_t,NULL);

	hipMalloc(&d_A, size);
	hipMalloc(&d_B, size);
	hipMalloc(&d_C, size);

	//copy memory
	hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice);
	hipMemcpy(d_B, h_B, size, hipMemcpyHostToDevice);

	//setup launch configuration
	dim3 threadsPerBlock(N,N,1);
	dim3 blocksPerGrid(1,1,1);
  threadsPerBlock.x = 32;
  threadsPerBlock.y = 32;
  blocksPerGrid.x = ceil(double(N)/(double)threadsPerBlock.x);
  blocksPerGrid.y = ceil(double(N)/(double)threadsPerBlock.y);

	gettimeofday(&start_c,NULL);

	//kernel call
	lib_mmul(d_A, d_B, d_C);

  hipDeviceSynchronize();

	gettimeofday(&end_c,NULL);

	//copy back
	hipMemcpy(h_C, d_C, size, hipMemcpyDeviceToHost);
	hipDeviceSynchronize();

	gettimeofday(&end_t,NULL);

  time_calc = ((end_c.tv_sec-start_c.tv_sec)*1.0)+ ((end_c.tv_usec - start_c.tv_usec) / 1000000.0);
  time_total = ((end_t.tv_sec-start_t.tv_sec)*1.0)+ ((end_t.tv_usec - start_t.tv_usec) / 1000000.0);

	//check by sum
	long double exp_sum = (long double)N;
	exp_sum = pow(exp_sum,3.0)*pow(exp_sum-1,2.0)/2.0;

	long double calc_sum = 0;
  for(i = 0; i < N; i++){
    for(j = 0; j < N; j++){
      calc_sum = calc_sum + h_C[N*i+j];
    }
  }

  printf("\nTesting %d x %d Matrix:\n",N,N);
  printf("------------------------------\n");
  printf("Expected Sum:        %Lf\n",exp_sum);
  printf("Calculated Sum:      %Lf\n",calc_sum);
  printf("------------------------------\n");
  printf("Total Time:          %f\n",time_total);  
  printf("Calculation Time:    %f\n\n",time_calc);

	//cleanup
	hipFree(d_A);
	hipFree(d_B);
	hipFree(d_C);

	free(h_A);
	free(h_B);
	free(h_C);

	return hipDeviceSynchronize();
}


